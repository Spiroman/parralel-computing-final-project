#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <iostream>
#include <stddef.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "data.h"
#include "cudaFunctions.h"

#define MAX_THREADS 256
#define WEIGHTS 4
// #define DEBUG 1
// #define DEBUG_FINAL 1

__device__ char conservativeGroup[GROUP_A_ROWS][GROUP_A_COLS] = {
    "NDEQ",
    "MILV",
    "FYW",
    "NEQK",
    "QHRK",
    "HY",
    "STA",
    "NHQK",
    "MILF"};

__device__ char semiConservativeGroup[GROUP_B_ROWS][GROUP_B_COLS] = {
    "SAG",
    "SGND",
    "NEQHRK",
    "HFY",
    "ATV",
    "STPA",
    "NDEQHK",
    "FVLIM",
    "CSA",
    "STNK",
    "SNDEQK"};

__device__ int checkConservativeGroup(char seq1, char seq2)
{

    for (int i = 0; i < GROUP_A_ROWS; i++)
    {
        for (int j = 0; j < GROUP_A_COLS; j++)
        {
            if ((conservativeGroup[i][j] == seq1) && (conservativeGroup[i][j] == seq2))
                return 1;
        }
    }
    return 0;
}

__device__ int checkSemiConservativeGroup(char seq1, char seq2)
{
    for (int i = 0; i < GROUP_A_ROWS; i++)
    {
        for (int j = 0; j < GROUP_A_COLS; j++)
        {
            if ((semiConservativeGroup[i][j] == seq1) && (semiConservativeGroup[i][j] == seq2))
                return 1;
        }
    }
    return 0;
}

__global__ void determinePartialScores(char *baseSeq, char *mutation, int *cmpRes, int *weights, int numOfChecks)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= numOfChecks)
        return;
    if (tid < numOfChecks)
    {
        #ifdef DEBUG
        // Check all passed parameters and their assignment
        printf("base: ");
        for(int i =0; i< numOfChecks; i++){
            printf("%c", baseSeq[i]);
        }
        printf(" . mutation: ");
        for(int i =0; i< numOfChecks; i++){
            printf("%c", mutation[i]);
        }
        printf(" weights:");
        for(int i = 0; i<4; i++){
            printf("%d", weights[i]);
        }
	    // printf("tid: %d, base:%c, mut: %c\n", tid, baseSeq[tid], mutation[tid]);
        #endif

        // For each type of match/missmatch we will assign the score of the match directly instead of the the char.
        // Meaning: if we have a full match, we will assign the weight of the full match in the result array
        // instead of putting '*', which will make the final calculation quicker (less checks for the type of char in each index)
        
        if (baseSeq[tid] == mutation[tid])
        {
            // Complete match -> '*' in our assignment
            cmpRes[tid] = weights[0];
        }
        else if (checkConservativeGroup(baseSeq[tid],mutation[tid]))
        {
            // Conservative match -> ':' in our assignment
            cmpRes[tid] = weights[1];
        }
        else if (checkSemiConservativeGroup(baseSeq[tid], mutation[tid]))
        {
            // Semi-Conservative match -> '.' in our assignment
            cmpRes[tid] = weights[2];
        }
        else
        {
            // Not a match -> ' ' in our assignment
            cmpRes[tid] = weights[3];
        }

        #ifdef DEBUG
        // Check result in device
        printf(" cmp: ");
        for (int i = 0; i < numOfChecks; i++)
        {
            printf("%d,", cmpRes[i]);
        }
        printf("\n");
        #endif
    }
}

void checkError(hipError_t hipError_t, const char* s_err)
{
    if (hipError_t != hipSuccess)
    {
        fprintf(stderr, "%s - %s\n", s_err, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }
}

void launchCuda(char *baseSeq, char *mutation, int lenOfAugmented, int *cmpRes, int *weights)
{
    // Error code to check return values for CUDA calls
    hipError_t hipError_t = hipSuccess;
    char *cuda_baseSeq;
    char *cuda_mutation;
    int *cuda_cmpRes;
    int *cuda_weights;
    
    // Allocate memory on GPU
    hipError_t = hipMalloc((void **)&cuda_baseSeq, lenOfAugmented);
    checkError(hipError_t, "Failed to allocate device memory seq2");

    hipError_t = hipMalloc((void **)&cuda_mutation, lenOfAugmented);
    checkError(hipError_t, "Failed to allocate device memory seq1");

    hipError_t = hipMalloc((void **)&cuda_cmpRes, lenOfAugmented * sizeof(int));
    checkError(hipError_t, "Failed to allocate device memory w_cuda-");

    hipError_t = hipMalloc((void **)&cuda_weights, WEIGHTS * sizeof(int));
    checkError(hipError_t, "Failed to allocate device memory w_cuda-");

    // Copy from host to device
    hipError_t = hipMemcpy(cuda_baseSeq, baseSeq, lenOfAugmented, hipMemcpyHostToDevice);
    checkError(hipError_t, "Failed to copy data from host to device seq2 -");

    hipError_t = hipMemcpy(cuda_mutation, mutation, lenOfAugmented, hipMemcpyHostToDevice);
    checkError(hipError_t, "Failed to copy data from host to device seq1 -");

    hipError_t = hipMemcpy(cuda_weights, weights, WEIGHTS * sizeof(int), hipMemcpyHostToDevice);
    checkError(hipError_t, "Failed to copy data from host to device w_cuda -");

    // Calculate the number of blocks
    int blocksPerGrid = (lenOfAugmented + MAX_THREADS - 1) / MAX_THREADS;
    
    #ifdef DEBUG_FINAL
    // printf("The number of blocks per grid is: %d\n", blocksPerGrid);
    printf("mutation: ");
    for (int i = 0; i < lenOfAugmented; i++)
    {
        printf("%c", mutation[i]);
    }
    printf("\n");
    #endif

    // Launch the Kernel
    determinePartialScores<<<blocksPerGrid, MAX_THREADS>>>(cuda_baseSeq, cuda_mutation, cuda_cmpRes, cuda_weights, lenOfAugmented);
    hipError_t = hipDeviceSynchronize();
    checkError(hipError_t, "Failed to synch kernel -");

    hipError_t = hipGetLastError();
    checkError(hipError_t, "Failed kernel -");
    
    // Copy results
    hipError_t = hipMemcpy(cmpRes, cuda_cmpRes, lenOfAugmented * sizeof(int), hipMemcpyDeviceToHost);
    checkError(hipError_t, "Failed to copy data device to host results -");
    
    #ifdef DEBUG
    printf(" cmp: ");
    for (int i = 0; i < lenOfAugmented; i++)
    {
        printf("%d,", cmpRes[i]);
    }
    printf("\n");
    printf("tid: %d, base:%c, mut: %c\n", tid, baseSeq[tid], mutation[tid]);
    #endif

    hipError_t = hipFree(cuda_baseSeq);
    checkError(hipError_t, "Failed to free base sequence -");

    hipError_t = hipFree(cuda_mutation);
    checkError(hipError_t, "Failed to free mutation sequence -");

    hipError_t = hipFree(cuda_cmpRes);
    checkError(hipError_t, "Failed to free results -");   

    hipError_t = hipFree(cuda_weights);
    checkError(hipError_t, "Failed to free weights -");
    
    return;
}


